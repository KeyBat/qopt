
#include <hip/hip_runtime.h>
__constant__ int nreal;
__constant__ int nfunc;

// wszystko ponizsze, co jest stale, chyba powinno byc w constant (wspolne dla wszystkich watkow)
__device__ double C;
__device__ double global_bias;
__device__ double *trans_x; // RW (te chyba powinny byc w shared, ale osobne dla kazdego watku)
__device__ double *temp_x1; // RW
__device__ double *temp_x2; // RW
__device__ double *temp_x3; // RW
__device__ double *temp_x4; // RW
__device__ double *norm_x;  // RW
__device__ double *basic_f; // nfunc  RW
__device__ double *weight; // nfunc   RW
__device__ double *sigma; // nfunc
__device__ double *lambda; // nfunc
__device__ double *bias; // nfunc
__device__ double *norm_f; // nfunc  RW
__device__ double **o;
__device__ double **g;
__device__ double ***l;

__device__ double *l_flat;

__device__ double calc_sphere (double *x)
{
    int i;
    double res;
    res = 0.0;
    for (i=0; i<nreal; i++)
    {
        res += x[i]*x[i];
    }
    return (res);
}

__device__ double calc_schwefel (double *x)
{
    int i, j;
    double sum1, sum2;
    sum1 = 0.0;
    for (i=0; i<nreal; i++)
    {
        sum2 = 0.0;
        for (j=0; j<=i; j++)
        {
            sum2 += x[j];
        }
        sum1 += sum2*sum2;
    }
    return (sum1);
}

__device__ void transform (double  * x, int count)
{
    int i, j;
    for (i=0; i<nreal; i++)
    {
        temp_x1[i] = x[i] - o[count][i];
    }
    for (i=0; i<nreal; i++)
    {
        temp_x2[i] = temp_x1[i]/lambda[count];
    }
    for (j=0; j<nreal; j++)
    {
        temp_x3[j] = 0.0;
        for (i=0; i<nreal; i++)
        {
            temp_x3[j] += g[i][j]*temp_x2[i];
        }
    }
    for (j=0; j<nreal; j++)
    {
        trans_x[j] = 0.0;
        for (i=0; i<nreal; i++)
        {
            // trans_x[j] += l[count][i][j]*temp_x3[i];
            trans_x[j] += l_flat[count * (nreal * nreal) + i * nreal + j] *temp_x3[i];
        }
    }
    return;
}

// F1
__global__ void calc_benchmark_func_f1(double *x, double *res)
{
    transform (x, 0);
    basic_f[0] = calc_sphere (trans_x);
    res[0] = basic_f[0] + bias[0];
}

// F2
__global__ void calc_benchmark_func_f2(double *x, double *res)
{
    transform (x, 0);
    basic_f[0] = calc_schwefel (trans_x);
    res[0] = basic_f[0] + bias[0];
}

// F3
__global__ void calc_benchmark_func_f3(double *x, double *res)
{
    int i;
    transform (x, 0);
    basic_f[0] = 0.0;
    for (i=0; i<nreal; i++)
    {
        basic_f[0] += trans_x[i]*trans_x[i]*pow(1.0e6,i/(nreal-1.0));
    }
    res[0] = basic_f[0] + bias[0];
}

__global__ void f(double *arg, double *result)
{
    result[threadIdx.x] = calc_sphere(arg);
    //result[threadIdx.x] = 1;
}

__global__ void test(double *result, double *o_out, double *g_out, double *l_out) {
    //MEM = *foo;
    //result[0] = sizeof(MEM_t);
    result[0] = l_flat[0];
    result[1] = C;
    result[2] = trans_x[0];
    result[3] = temp_x4[1];
    result[4] = norm_x[0];
    result[5] = norm_f[0];
    result[6] = o[1][0];
    result[7] = bias[0];

    for (int i = 0; i < nfunc; i++) {
        for (int j = 0; j < nreal; j++) {
            o_out[i * nreal + j] = o[i][j];
        }
    }

    for (int i = 0; i < nreal; i++) {
        for (int j = 0; j < nreal; j++) {
            g_out[i * nreal + j] = g[i][j];
        }
    }

    l_out[0] = 13;

    for (int i = 0; i < nfunc * nreal * nreal; i++) {
        l_out[i] = l_flat[i];
    }
}
